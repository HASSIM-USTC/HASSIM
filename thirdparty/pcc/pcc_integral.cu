#include "hip/hip_runtime.h"
/*
 * Doing the 4 dimensional integration in Photon coincidence counting
 */
#include <cstdio>
#include <cstdlib>
#include <thrust/complex.h>
#include "json11.hpp"

using namespace std;
using namespace json11;

const complex<double> _ci = complex<double>(0.0,1.0);

__device__ inline double hstep (const double x) {
    if (x < 0) 
        return 0;
    else if (x > 0) 
        return 1.0;
    return 0.5;
}

__device__ inline complex<double> Dtw (const double tpr, const double tau, const double t, const double w, const double st, const double sw) {
    return 1.0/(2*sw)*hstep(tpr-t)*hstep(tpr+tau-t)*exp(-(_ci*w+st)*tau-2*st*(tpr-t))*(hstep(tau)*exp(-sw*tau)+hstep(-tau)*exp(sw*tau));
}

__device__ inline int at (const int nt, const int it0, const int it1, const int it2, const int it3) {
    return nt*(nt*(nt*it3+it2)+it1)+it0;
} 

__global__ void pccKernel (double* s2, const complex<double>* ft1, const complex<double>* ft2, const int nt, const int nw, const double tf, const double wi, const double wf, const double ws, const double t1, const double t2, const double st1, const double sw1, const double st2, const double sw2) {

    const int bx = blockIdx.x;
    const int tx = threadIdx.x;

    const double dt = tf/nt;
    const double dw = (wf-wi)/nw;
    const double dt4 = dt*dt*dt*dt; 
    const double w1 = wi+dw*bx;
    const double w2 = wi+dw*tx;
    complex<double> signal = 0;
    for (int it0=0; it0<nt; ++it0) { 
    for (int it1=0; it1<nt; ++it1) {
    for (int it2=0; it2<nt; ++it2) {
    for (int it3=0; it3<nt; ++it3) {
        const int loc = at(nt,it0,it1,it2,it3);
        {
            double tau2 = dt*it1;
            double t2pr = dt*it0;
            double t1pr = t2pr+tau2+dt*it2;
            double tau1 = dt*it3;
            singal += Dtw(t1pr,tau1,t1,w1,st1,sw1)*Dtw(t2pr,tau2,t2,w2,st2,sw2)*ft1[loc]*exp(_ci*ws*(tau1+tau2));
        }
        {
            double tau2 =-dt*it1;
            double t2pr = dt*it0-tau2;
            double t1pr = dt*it2+t2pr;
            double tau1 = dt*it3;
            signal += Dtw(t1pr,tau1,t1,w1,st1,sw1)*Dtw(t2pr,tau2,t2,w2,st2,sw2)*ft2[loc]*exp(_ci*ws*(tau1+tau2));
        }
    }}}}
    s2[bx*blockDim.x+tx] = real(signal)*dt4;
}


int main () {

    ifstream jsonFile("input_pcc.json");
    stringstream strStream;
    strStream << jsonFile.rdbuf();
    string jsonStr = strStream.str();
    string err;

    const Json json = Json::parse(jsonStr,err);
    if (!err.empty()) {
        printf ("Error in parsing input file: %s\n", err.c_str());
        return 0;
    } 
    
    const int    nt = json["nt"].int_value();
    const int    nw = json["nw"].int_value();
    const double tf = json["tf"].number_value();
    const double wi = json["wi"].number_value();
    const double wf = json["wf"].number_value();
    const double ws = json["ws"].number_value();
    const double t1 = json["t1"].number_value();
    const double t2 = json["t2"].number_value();
    const double st1 = json["st1"].number_value();
    const double sw1 = json["sw1"].number_value();
    const double st2 = json["st2"].number_value();
    const double sw2 = json["sw2"].number_value();
    
    const double dt = tf/nt;
    const double dw = (wf-wi)/nw;
    const double dt4 = dt*dt*dt*dt; 
    const int nt4 = nt*nt*nt*nt; 
    
    complex<double> *ft1, *ft2;
    double *s2;
    hipMallocManaged(&ft1, nt4*sizeof(complex<double>));
    hipMallocManaged(&ft2, nt4*sizeof(complex<double>));
    hipMallocManaged( &s2, nt4*sizeof(double));

    double re(0), im(0);
    FILE *fs1 = fopen("barePcc_1.dat","r");
    FILE *fs2 = fopen("barePcc_2.dat","r");
    for (int i=0; i<nt4; ++i) {
        fscanf(fs1,"%lf",&re,&im);
        fscanf(fs2,"%lf",&re,&im);
    }

    pccKernel<<<nw,nw>>>(s2, ft1, ft2, nt, nw, tf, wi, wf, ws, t1, t2, st1, sw1, st2, sw2);

    FILE *fs = fopen("s2.mat","w");
    for (int iw1=0; iw1<nw; ++iw1) {
        for (int iw2=0; iw2<nw; ++iw2) {
            fprintf (fs, "%16.6e", s2[iw1*nw+iw2]);
        }
        fprintf (fs, "\n");
    }
    fclose(fs);
    
    hipFree(s2);
    hipFree(ft1);
    hipFree(ft2);

    return 0;
}
